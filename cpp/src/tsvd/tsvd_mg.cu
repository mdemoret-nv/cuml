/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/common/device_buffer.hpp>
#include <cuml/decomposition/sign_flip_mg.hpp>
#include <cuml/decomposition/tsvd.hpp>
#include <cuml/decomposition/tsvd_mg.hpp>
#include <opg/linalg/mm_aTa.hpp>
#include <opg/stats/mean.hpp>
#include <opg/stats/mean_center.hpp>
#include <opg/stats/stddev.hpp>
#include <raft/comms/comms.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/eltwise.cuh>
#include <raft/matrix/math.cuh>
#include <raft/stats/mean_center.cuh>
#include "tsvd.cuh"

using namespace MLCommon;

namespace ML {
namespace TSVD {
namespace opg {

template <typename T>
void fit_impl(raft::handle_t &handle,
              std::vector<Matrix::Data<T> *> &input_data,
              Matrix::PartDescriptor &input_desc, T *components,
              T *singular_vals, paramsTSVD prms, hipStream_t *streams,
              int n_streams, bool verbose) {
  const auto &comm = handle.get_comms();
  hipblasHandle_t cublas_handle = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();

  // This variable should be updated to use `size_t`
  // Reference issue https://github.com/rapidsai/cuml/issues/2459
  int len = prms.n_cols * prms.n_cols;

  device_buffer<T> cov_data(allocator, streams[0], len);
  size_t cov_data_size = cov_data.size();
  Matrix::Data<T> cov{cov_data.data(), cov_data_size};

  LinAlg::opg::mm_aTa(handle, cov, input_data, input_desc, streams, n_streams);

  device_buffer<T> components_all(allocator, streams[0], len);
  device_buffer<T> explained_var_all(allocator, streams[0], prms.n_cols);

  ML::calEig(handle, cov.ptr, components_all.data(), explained_var_all.data(),
             prms, streams[0]);

  raft::matrix::truncZeroOrigin(components_all.data(), prms.n_cols, components,
                                prms.n_components, prms.n_cols, streams[0]);

  T scalar = T(1);
  raft::matrix::seqRoot(explained_var_all.data(), singular_vals, scalar,
                        prms.n_components, streams[0]);
}

/**
 * @brief      performs MNMG fit operation for the tsvd
 *
 * @param[in]  handle         the internal cuml handle object
 * @param[in]  rank_sizes     includes all the partition size information for
 *                            the rank
 * @param[in]  n_parts        number of partitions
 * @param[in]  input          input data
 * @param[out] components     principal components of the input data
 * @param[out] singular_vals  singular values of the data
 * @param[in]  prms           data structure that includes all the parameters
 *                            from input size to algorithm
 * @param[in]  verbose        The verbose
 *
 * @tparam     T              { description }
 */
template <typename T>
void fit_impl(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
              size_t n_parts, Matrix::Data<T> **input, T *components,
              T *singular_vals, paramsTSVD prms, bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);

  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);
  Matrix::PartDescriptor input_desc(prms.n_rows, prms.n_cols, ranksAndSizes,
                                    rank);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  fit_impl(handle, input_data, input_desc, components, singular_vals, prms,
           streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

template <typename T>
void transform_impl(raft::handle_t &handle,
                    std::vector<Matrix::Data<T> *> &input,
                    Matrix::PartDescriptor input_desc, T *components,
                    std::vector<Matrix::Data<T> *> &trans_input,
                    paramsTSVD prms, hipStream_t *streams, int n_streams,
                    bool verbose) {
  int rank = handle.get_comms().get_rank();

  hipblasHandle_t hipblas.h = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();

  std::vector<Matrix::RankSizePair *> local_blocks =
    input_desc.blocksOwnedBy(rank);

  for (int i = 0; i < input.size(); i++) {
    int si = i % n_streams;

    T alpha = T(1);
    T beta = T(0);
    raft::linalg::gemm(handle, input[i]->ptr, local_blocks[i]->size,
                       size_t(prms.n_cols), components, trans_input[i]->ptr,
                       local_blocks[i]->size, int(prms.n_components),
                       HIPBLAS_OP_N, HIPBLAS_OP_T, alpha, beta, streams[si]);
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }
}

/**
 * @brief      performs MNMG transform operation for the tsvd.
 *
 * @param[in]  handle       the internal cuml handle object
 * @param[in]  rank_sizes   includes all the partition size information for the
 *                          rank
 * @param[in]  n_parts      number of partitions
 * @param[in]  input        input data
 * @param[in]  components   principal components of the input data
 * @param[out] trans_input  transformed input data
 * @param[in]  prms         data structure that includes all the parameters from
 *                          input size to algorithm
 * @param[in]  verbose      The verbose
 *
 * @tparam     T            { description }
 */
template <typename T>
void transform_impl(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
                    size_t n_parts, Matrix::Data<T> **input, T *components,
                    Matrix::Data<T> **trans_input, paramsTSVD prms,
                    bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);
  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);
  Matrix::PartDescriptor input_desc(prms.n_rows, prms.n_cols, ranksAndSizes,
                                    rank);
  std::vector<Matrix::Data<T> *> trans_data(trans_input, trans_input + n_parts);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  transform_impl(handle, input_data, input_desc, components, trans_data, prms,
                 streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

template <typename T>
void inverse_transform_impl(raft::handle_t &handle,
                            std::vector<Matrix::Data<T> *> &trans_input,
                            Matrix::PartDescriptor trans_input_desc,
                            T *components,
                            std::vector<Matrix::Data<T> *> &input,
                            paramsTSVD prms, hipStream_t *streams,
                            int n_streams, bool verbose) {
  hipblasHandle_t hipblas.h = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();
  std::vector<Matrix::RankSizePair *> local_blocks =
    trans_input_desc.partsToRanks;

  for (int i = 0; i < local_blocks.size(); i++) {
    int si = i % n_streams;
    T alpha = T(1);
    T beta = T(0);

    raft::linalg::gemm(handle, trans_input[i]->ptr, local_blocks[i]->size,
                       size_t(prms.n_components), components, input[i]->ptr,
                       local_blocks[i]->size, prms.n_cols, HIPBLAS_OP_N,
                       HIPBLAS_OP_N, alpha, beta, streams[si]);
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }
}

/**
 * @brief      performs MNMG inverse transform operation for the output.
 *
 * @param[in]  handle       the internal cuml handle object
 * @param[in]  rank_sizes   includes all the partition size information for the
 *                          rank
 * @param[in]  n_parts      number of partitions
 * @param[in]  trans_input  transformed input data
 * @param[in]  components   principal components of the input data
 * @param[out] input        input data
 * @param[in]  prms         data structure that includes all the parameters from
 *                          input size to algorithm
 * @param[in]  verbose      The verbose
 *
 * @tparam     T            { description }
 */
template <typename T>
void inverse_transform_impl(raft::handle_t &handle,
                            Matrix::RankSizePair **rank_sizes, size_t n_parts,
                            Matrix::Data<T> **trans_input, T *components,
                            Matrix::Data<T> **input, paramsTSVD prms,
                            bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);
  Matrix::PartDescriptor trans_desc(prms.n_rows, prms.n_components,
                                    ranksAndSizes, rank);
  std::vector<Matrix::Data<T> *> trans_data(trans_input, trans_input + n_parts);

  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  inverse_transform_impl(handle, trans_data, trans_desc, components, input_data,
                         prms, streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

/**
 * @brief      performs MNMG fit and transform operation for the tsvd.
 *
 * @param[in]  handle               the internal cuml handle object
 * @param[in]  input_data           input data
 * @param      input_desc           The input description
 * @param[out] trans_data           transformed input data
 * @param[in]  trans_desc           includes all the partition size information
 *                                  for the rank
 * @param[out] components           principal components of the input data
 * @param[out] explained_var        explained var
 * @param[out] explained_var_ratio  the explained var ratio
 * @param[out] singular_vals        singular values of the data
 * @param[in]  prms                 data structure that includes all the
 *                                  parameters from input size to algorithm
 * @param[in]  verbose              The verbose
 *
 * @tparam     T                    { description }
 */
template <typename T>
void fit_transform_impl(raft::handle_t &handle,
                        std::vector<Matrix::Data<T> *> &input_data,
                        Matrix::PartDescriptor &input_desc,
                        std::vector<Matrix::Data<T> *> &trans_data,
                        Matrix::PartDescriptor &trans_desc, T *components,
                        T *explained_var, T *explained_var_ratio,
                        T *singular_vals, paramsTSVD prms, bool verbose) {
  int rank = handle.get_comms().get_rank();

  // TODO: These streams should come from raft::handle_t
  int n_streams = input_desc.blocksOwnedBy(rank).size();
  ;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  fit_impl(handle, input_data, input_desc, components, singular_vals, prms,
           streams, n_streams, verbose);

  transform_impl(handle, input_data, input_desc, components, trans_data, prms,
                 streams, n_streams, verbose);

  PCA::opg::sign_flip(handle, trans_data, input_desc, components,
                      prms.n_components, streams, n_streams);

  device_buffer<T> mu_trans(handle.get_device_allocator(), streams[0],
                            prms.n_components);
  Matrix::Data<T> mu_trans_data{mu_trans.data(), size_t(prms.n_components)};

  Stats::opg::mean(handle, mu_trans_data, trans_data, trans_desc, streams,
                   n_streams);

  Matrix::Data<T> explained_var_data{explained_var, size_t(prms.n_components)};

  Stats::opg::var(handle, explained_var_data, trans_data, trans_desc,
                  mu_trans_data.ptr, streams, n_streams);

  device_buffer<T> mu(handle.get_device_allocator(), streams[0], prms.n_rows);
  Matrix::Data<T> mu_data{mu.data(), size_t(prms.n_rows)};

  Stats::opg::mean(handle, mu_data, input_data, input_desc, streams, n_streams);

  device_buffer<T> var_input(handle.get_device_allocator(), streams[0],
                             prms.n_rows);
  Matrix::Data<T> var_input_data{var_input.data(), size_t(prms.n_rows)};

  Stats::opg::var(handle, var_input_data, input_data, input_desc, mu_data.ptr,
                  streams, n_streams);

  device_buffer<T> total_vars(handle.get_device_allocator(), streams[0], 1);
  raft::stats::sum(total_vars.data(), var_input_data.ptr, 1, prms.n_cols, false,
                   streams[0]);

  T total_vars_h;
  raft::update_host(&total_vars_h, total_vars.data(), 1, streams[0]);
  CUDA_CHECK(hipStreamSynchronize(streams[0]));
  T scalar = T(1) / total_vars_h;

  raft::linalg::scalarMultiply(explained_var_ratio, explained_var, scalar,
                               prms.n_components, streams[0]);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

void fit(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
         size_t n_parts, Matrix::floatData_t **input, float *components,
         float *singular_vals, paramsTSVD prms, bool verbose) {
  fit_impl(handle, rank_sizes, n_parts, input, components, singular_vals, prms,
           verbose);
}

void fit(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
         size_t n_parts, Matrix::doubleData_t **input, double *components,
         double *singular_vals, paramsTSVD prms, bool verbose) {
  fit_impl(handle, rank_sizes, n_parts, input, components, singular_vals, prms,
           verbose);
}

void fit_transform(raft::handle_t &handle,
                   std::vector<Matrix::Data<float> *> &input_data,
                   Matrix::PartDescriptor &input_desc,
                   std::vector<Matrix::Data<float> *> &trans_data,
                   Matrix::PartDescriptor &trans_desc, float *components,
                   float *explained_var, float *explained_var_ratio,
                   float *singular_vals, paramsTSVD prms, bool verbose) {
  fit_transform_impl(handle, input_data, input_desc, trans_data, trans_desc,
                     components, explained_var, explained_var_ratio,
                     singular_vals, prms, verbose);
}

void fit_transform(raft::handle_t &handle,
                   std::vector<Matrix::Data<double> *> &input_data,
                   Matrix::PartDescriptor &input_desc,
                   std::vector<Matrix::Data<double> *> &trans_data,
                   Matrix::PartDescriptor &trans_desc, double *components,
                   double *explained_var, double *explained_var_ratio,
                   double *singular_vals, paramsTSVD prms, bool verbose) {
  fit_transform_impl(handle, input_data, input_desc, trans_data, trans_desc,
                     components, explained_var, explained_var_ratio,
                     singular_vals, prms, verbose);
}

void transform(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
               size_t n_parts, Matrix::Data<float> **input, float *components,
               Matrix::Data<float> **trans_input, paramsTSVD prms,
               bool verbose) {
  transform_impl(handle, rank_sizes, n_parts, input, components, trans_input,
                 prms, verbose);
}

void transform(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
               size_t n_parts, Matrix::Data<double> **input, double *components,
               Matrix::Data<double> **trans_input, paramsTSVD prms,
               bool verbose) {
  transform_impl(handle, rank_sizes, n_parts, input, components, trans_input,
                 prms, verbose);
}

void inverse_transform(raft::handle_t &handle,
                       Matrix::RankSizePair **rank_sizes, size_t n_parts,
                       Matrix::Data<float> **trans_input, float *components,
                       Matrix::Data<float> **input, paramsTSVD prms,
                       bool verbose) {
  inverse_transform_impl(handle, rank_sizes, n_parts, trans_input, components,
                         input, prms, verbose);
}

void inverse_transform(raft::handle_t &handle,
                       Matrix::RankSizePair **rank_sizes, size_t n_parts,
                       Matrix::Data<double> **trans_input, double *components,
                       Matrix::Data<double> **input, paramsTSVD prms,
                       bool verbose) {
  inverse_transform_impl(handle, rank_sizes, n_parts, trans_input, components,
                         input, prms, verbose);
}

}  // namespace opg
}  // namespace TSVD
}  // namespace ML
